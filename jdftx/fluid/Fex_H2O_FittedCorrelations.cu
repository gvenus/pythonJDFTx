#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------
Copyright 2011 Ravishankar Sundararaman

This file is part of JDFTx.

JDFTx is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

JDFTx is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with JDFTx.  If not, see <http://www.gnu.org/licenses/>.
-------------------------------------------------------------------*/

#include <core/GpuKernelUtils.h>
#include <fluid/Fex_H2O_FittedCorrelations_internal.h>

__global__
void Fex_H20_FittedCorrelations_kernel(int nr, const double* NObar, const double* NHbar,
	double* Fex, double* Phi_NObar, double* Phi_NHbar)
{	int i = kernelIndex1D();
	if(i<nr)
		Fex[i] = Fex_H2O_FittedCorrelations_calc(i, NObar, NHbar, Phi_NObar, Phi_NHbar);
}
void Fex_H20_FittedCorrelations_gpu(int nr, const double* NObar, const double* NHbar,
	double* Fex, double* Phi_NObar, double* Phi_NHbar)
{	GpuLaunchConfig1D glc(Fex_H20_FittedCorrelations_kernel, nr);
	Fex_H20_FittedCorrelations_kernel<<<glc.nBlocks, glc.nPerBlock>>>(nr, NObar, NHbar, Fex, Phi_NObar, Phi_NHbar);
}

