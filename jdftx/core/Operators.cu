#include "hip/hip_runtime.h"
/*-------------------------------------------------------------------
Copyright 2011 Ravishankar Sundararaman

This file is part of JDFTx.

JDFTx is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

JDFTx is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with JDFTx.  If not, see <http://www.gnu.org/licenses/>.
-------------------------------------------------------------------*/

#include <core/GpuKernelUtils.h>
#include <core/LoopMacros.h>
#include <core/Operators_internal.h>


__global__
void RealG_kernel(int zBlock, const vector3<int> S, const complex* vFull, complex* vHalf, double scaleFac)
{	COMPUTE_halfGindices
	RealG_calc(i, iG, S, vFull, vHalf, scaleFac);
}
void RealG_gpu(const vector3<int> S, const complex* vFull, complex* vHalf, double scaleFac)
{	GpuLaunchConfigHalf3D glc(RealG_kernel, S);
	for(int zBlock=0; zBlock<glc.zBlockMax; zBlock++)
		RealG_kernel<<<glc.nBlocks,glc.nPerBlock>>>(zBlock, S, vFull, vHalf, scaleFac);
	gpuErrorCheck();
}

__global__
void ImagG_kernel(int zBlock, const vector3<int> S, const complex* vFull, complex* vHalf, double scaleFac)
{	COMPUTE_halfGindices
	ImagG_calc(i, iG, S, vFull, vHalf, scaleFac);
}
void ImagG_gpu(const vector3<int> S, const complex* vFull, complex* vHalf, double scaleFac)
{	GpuLaunchConfigHalf3D glc(ImagG_kernel, S);
	for(int zBlock=0; zBlock<glc.zBlockMax; zBlock++)
		ImagG_kernel<<<glc.nBlocks,glc.nPerBlock>>>(zBlock, S, vFull, vHalf, scaleFac);
	gpuErrorCheck();
}

__global__
void ComplexG_kernel(int zBlock, const vector3<int> S, const complex* vHalf, complex *vFull, double scaleFac)
{	COMPUTE_halfGindices
	ComplexG_calc(i, iG, S, vHalf, vFull, scaleFac);
}
void ComplexG_gpu(const vector3<int> S, const complex* vHalf, complex *vFull, double scaleFac)
{	GpuLaunchConfigHalf3D glc(ComplexG_kernel, S);
	for(int zBlock=0; zBlock<glc.zBlockMax; zBlock++)
		ComplexG_kernel<<<glc.nBlocks,glc.nPerBlock>>>(zBlock, S, vHalf, vFull, scaleFac);
	gpuErrorCheck();
}



__global__
void L_kernel(int zBlock, const vector3<int> S, const matrix3<> GGT, complex* v)
{	COMPUTE_halfGindices
	v[i] *= GGT.metric_length_squared(iG);
}
void L_gpu(const vector3<int> S, const matrix3<> GGT, complex* v)
{	GpuLaunchConfigHalf3D glc(L_kernel, S);
	for(int zBlock=0; zBlock<glc.zBlockMax; zBlock++)
		L_kernel<<<glc.nBlocks,glc.nPerBlock>>>(zBlock, S, GGT, v);
	gpuErrorCheck();
}

__global__
void Linv_kernel(int zBlock, const vector3<int> S, const matrix3<> GGT, complex* v)
{	COMPUTE_halfGindices
	v[i] *= i ? 1.0/GGT.metric_length_squared(iG) : 0.0;
}
void Linv_gpu(const vector3<int> S, const matrix3<> GGT, complex* v)
{	GpuLaunchConfigHalf3D glc(Linv_kernel, S);
	for(int zBlock=0; zBlock<glc.zBlockMax; zBlock++)
		Linv_kernel<<<glc.nBlocks,glc.nPerBlock>>>(zBlock, S, GGT, v);
	gpuErrorCheck();
}

__global__
void fullL_kernel(int zBlock, const vector3<int> S, const matrix3<> GGT, complex* v)
{	COMPUTE_fullGindices
	v[i] *= GGT.metric_length_squared(iG);
}
void fullL_gpu(const vector3<int> S, const matrix3<> GGT, complex* v)
{	GpuLaunchConfig3D glc(fullL_kernel, S);
	for(int zBlock=0; zBlock<glc.zBlockMax; zBlock++)
		fullL_kernel<<<glc.nBlocks,glc.nPerBlock>>>(zBlock, S, GGT, v);
	gpuErrorCheck();
}

__global__
void fullLinv_kernel(int zBlock, const vector3<int> S, const matrix3<> GGT, complex* v)
{	COMPUTE_fullGindices
	v[i] *= i ? 1.0/GGT.metric_length_squared(iG) : 0.0;
}
void fullLinv_gpu(const vector3<int> S, const matrix3<> GGT, complex* v)
{	GpuLaunchConfig3D glc(fullLinv_kernel, S);
	for(int zBlock=0; zBlock<glc.zBlockMax; zBlock++)
		fullLinv_kernel<<<glc.nBlocks,glc.nPerBlock>>>(zBlock, S, GGT, v);
	gpuErrorCheck();
}



__global__
void exp_kernel(int N, double* X, double prefac)
{	int i = kernelIndex1D(); if(i<N) X[i] = exp(prefac*X[i]);
}
void exp_gpu(int N, double* X, double prefac)
{	GpuLaunchConfig1D glc(exp_kernel, N);
	exp_kernel<<<glc.nBlocks,glc.nPerBlock>>>(N, X, prefac);
	gpuErrorCheck();
}

__global__
void log_kernel(int N, double* X, double prefac)
{	int i = kernelIndex1D(); if(i<N) X[i] = log(prefac*X[i]);
}
void log_gpu(int N, double* X, double prefac)
{	GpuLaunchConfig1D glc(log_kernel, N);
	log_kernel<<<glc.nBlocks,glc.nPerBlock>>>(N, X, prefac);
	gpuErrorCheck();
}

__global__
void sqrt_kernel(int N, double* X, double prefac)
{	int i = kernelIndex1D(); if(i<N) X[i] = sqrt(prefac*X[i]);
}
void sqrt_gpu(int N, double* X, double prefac)
{	GpuLaunchConfig1D glc(sqrt_kernel, N);
	sqrt_kernel<<<glc.nBlocks,glc.nPerBlock>>>(N, X, prefac);
	gpuErrorCheck();
}

__global__
void inv_kernel(int N, double* X, double prefac)
{	int i = kernelIndex1D(); if(i<N) X[i] = prefac/X[i];
}
void inv_gpu(int N, double* X, double prefac)
{	GpuLaunchConfig1D glc(inv_kernel, N);
	inv_kernel<<<glc.nBlocks,glc.nPerBlock>>>(N, X, prefac);
	gpuErrorCheck();
}

__global__
void pow_kernel(int N, double* X, double scale, double alpha)
{	int i = kernelIndex1D(); if(i<N) X[i] = pow(scale*X[i],alpha);
}
void pow_gpu(int N, double* X, double scale, double alpha)
{	GpuLaunchConfig1D glc(pow_kernel, N);
	pow_kernel<<<glc.nBlocks,glc.nPerBlock>>>(N, X, scale, alpha);
	gpuErrorCheck();
}

__global__
void gaussConvolve_kernel(int zBlock, const vector3<int> S, const matrix3<> GGT, complex* data, double sigma)
{	COMPUTE_halfGindices
	data[i] *= exp(-0.5*sigma*sigma*GGT.metric_length_squared(iG));
}
void gaussConvolve_gpu(const vector3<int>& S, const matrix3<>& GGT, complex* data, double sigma)
{	GpuLaunchConfig3D glc(gaussConvolve_kernel, S);
	for(int zBlock=0; zBlock<glc.zBlockMax; zBlock++)
		gaussConvolve_kernel<<<glc.nBlocks,glc.nPerBlock>>>(zBlock, S, GGT, data, sigma);
}


__global__
void changeGrid_kernel(int zBlock, const vector3<int> S, const vector3<int> Sin, const vector3<int> Sout, const complex* in, complex* out)
{	COMPUTE_halfGindices
	changeGrid_calc(iG, Sin, Sout, in, out);
}
void changeGrid_gpu(const vector3<int>& S, const vector3<int>& Sin, const vector3<int>& Sout, const complex* in, complex* out)
{	GpuLaunchConfigHalf3D glc(changeGrid_kernel, S);
	for(int zBlock=0; zBlock<glc.zBlockMax; zBlock++)
		changeGrid_kernel<<<glc.nBlocks,glc.nPerBlock>>>(zBlock, S, Sin, Sout, in, out);
	gpuErrorCheck();
}

__global__
void changeGridFull_kernel(int zBlock, const vector3<int> S, const vector3<int> Sin, const vector3<int> Sout, const complex* in, complex* out)
{	COMPUTE_fullGindices
	changeGridFull_calc(iG, Sin, Sout, in, out);
}
void changeGridFull_gpu(const vector3<int>& S, const vector3<int>& Sin, const vector3<int>& Sout, const complex* in, complex* out)
{	GpuLaunchConfig3D glc(changeGridFull_kernel, S);
	for(int zBlock=0; zBlock<glc.zBlockMax; zBlock++)
		changeGridFull_kernel<<<glc.nBlocks,glc.nPerBlock>>>(zBlock, S, Sin, Sout, in, out);
	gpuErrorCheck();
}


__global__
void gradient_kernel(int zBlock, const vector3<int> S, const matrix3<> G, const complex* Xtilde, vector3<complex*> gradTilde)
{	COMPUTE_halfGindices
	gradient_calc(i, iG, IS_NYQUIST, G, Xtilde, gradTilde);
}
void gradient_gpu(const vector3<int> S, const matrix3<> G, const complex* Xtilde, vector3<complex*> gradTilde)
{	GpuLaunchConfigHalf3D glc(gradient_kernel, S);
	for(int zBlock=0; zBlock<glc.zBlockMax; zBlock++)
		gradient_kernel<<<glc.nBlocks,glc.nPerBlock>>>(zBlock, S, G, Xtilde, gradTilde);
	gpuErrorCheck();
}


__global__
void divergence_kernel(int zBlock, const vector3<int> S, const matrix3<> G, vector3<const complex*> Vtilde, complex* divTilde)
{	COMPUTE_halfGindices
	divergence_calc(i, iG, IS_NYQUIST, G, Vtilde, divTilde);
}
void divergence_gpu(const vector3<int> S, const matrix3<> G, vector3<const complex*> Vtilde, complex* divTilde)
{	GpuLaunchConfigHalf3D glc(divergence_kernel, S);
	for(int zBlock=0; zBlock<glc.zBlockMax; zBlock++)
		divergence_kernel<<<glc.nBlocks,glc.nPerBlock>>>(zBlock, S, G, Vtilde, divTilde);
	gpuErrorCheck();
}


__global__
void tensorGradient_kernel(int zBlock, const vector3<int> S, const matrix3<> G, const complex* Xtilde, tensor3<complex*> gradTilde)
{	COMPUTE_halfGindices
	tensorGradient_calc(i, iG, IS_NYQUIST, G, Xtilde, gradTilde);
}
void tensorGradient_gpu(const vector3<int> S, const matrix3<> G, const complex* Xtilde, tensor3<complex*> gradTilde)
{	GpuLaunchConfigHalf3D glc(tensorGradient_kernel, S);
	for(int zBlock=0; zBlock<glc.zBlockMax; zBlock++)
		tensorGradient_kernel<<<glc.nBlocks,glc.nPerBlock>>>(zBlock, S, G, Xtilde, gradTilde);
	gpuErrorCheck();
}


__global__
void tensorDivergence_kernel(int zBlock, const vector3<int> S, const matrix3<> G, tensor3<const complex*> Vtilde, complex* divTilde)
{	COMPUTE_halfGindices
	tensorDivergence_calc(i, iG, IS_NYQUIST, G, Vtilde, divTilde);
}
void tensorDivergence_gpu(const vector3<int> S, const matrix3<> G, tensor3<const complex*> Vtilde, complex* divTilde)
{	GpuLaunchConfigHalf3D glc(tensorDivergence_kernel, S);
	for(int zBlock=0; zBlock<glc.zBlockMax; zBlock++)
		tensorDivergence_kernel<<<glc.nBlocks,glc.nPerBlock>>>(zBlock, S, G, Vtilde, divTilde);
	gpuErrorCheck();
}


